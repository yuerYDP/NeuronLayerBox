#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

__device__ static void LTS(int *input,struct axon *neuro, unsigned char *spike,struct neuron_I *Ix, int number)
{
  //设置神经元计算参数
  float C=100;
  float k=1;
  float vr=-56;
  float vt=-42;
  float G_up=1;
  float G_down=1;
  float a=0.03;
  float b=8;
  float c=-50;
  float d=20;
  float v_peak=40;
  float sq=0,sp=0;
  float I;


  float v=neuro[number].v;
  float u=neuro[number].u;
  I=Ix[number].I;



  //Izhikevich model
     v=v+tau*(k*(v-vr)*(v-vt)-u+I)/C;
     u=u+tau*a*(b*(v-vr)-u);
     spike[number]=0;
     if(v>v_peak)
     {
       v=c;
       u=u+d;
       spike[number]=1;
     }
     u=fmin(670,u);

  neuro[number].v=v;
  neuro[number].u=u;
  Ix[number].I=0;
}

__global__ static void LTS_neuron(int *input,struct axon *neuro, unsigned char *spike,struct neuron_I *Ix, int *boxnum, int *THREAD_NUM, int *BLOCK_NUM)
{
const int tid = threadIdx.x;
const int bid = blockIdx.x;
int number=(THREAD_NUM[0]*BLOCK_NUM[0]+THREAD_NUM[1]*BLOCK_NUM[1]+THREAD_NUM[2]*BLOCK_NUM[2]+THREAD_NUM[3]*BLOCK_NUM[3]+THREAD_NUM[4]*BLOCK_NUM[4])*10+(bid * THREAD_NUM[5] + tid)*10;


/********第一个神经元虚拟计算内核*********/
if((number+0)<=boxnum[5])
{LTS(input,neuro,spike,Ix,number+0);}

/********第二个神经元虚拟计算内核********/
if((number+1)<=boxnum[5])
{LTS(input,neuro,spike,Ix,number+1);}

/********第三个神经元虚拟计算内核********/
if((number+2)<=boxnum[5])
{LTS(input,neuro,spike,Ix,number+2);}

/********第四个神经元虚拟计算内核*********/
if((number+3)<=boxnum[5])
{LTS(input,neuro,spike,Ix,number+3);}

/********第五个神经元虚拟计算内核*********/
if((number+4)<=boxnum[5])
{LTS(input,neuro,spike,Ix,number+4);}

/********第六个神经元虚拟计算内核*********/
if((number+5)<=boxnum[5])
{LTS(input,neuro,spike,Ix,number+5);}

/********第七个神经元虚拟计算内核********/
if((number+6)<=boxnum[5])
{LTS(input,neuro,spike,Ix,number+6);}

/********第八个神经元虚拟计算内核*********/
if((number+7)<=boxnum[5])
{LTS(input,neuro,spike,Ix,number+7);}

/********第九个神经元虚拟计算内核*********/
if((number+8)<=boxnum[5])
{LTS(input,neuro,spike,Ix,number+8);}

/********第十个神经元虚拟计算内核*********/
if((number+9)<=boxnum[5])
{LTS(input,neuro,spike,Ix,number+9);}

}
