#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
//neurontype:RS     example:layer2/3 layer4 layer5 layer6I layer6II


__device__ static void RS(int *input,struct axon *neuro, unsigned char *spike,struct neuron_I *Ix, int number)
{
  //设置神经元计算参数
  float C=100;
  float k=3;
  float vr=-60;
  float vt=-50;
  float G_up=3.0;
  float G_down=5.0;
  float a=0.01;
  float b=5;
  float c=-60;
  float d=400;
  float v_peak=50;
  float I;
  float I_distal;
  float I_proximal;

  float v=neuro[number].v;
  float u=neuro[number].u;
  I=Ix[number].I;


  //if(neuro[number].layer==5 && number==120000){printf("v2=%f,v1=%f,v0=%f,I_proximal=%f,I=%f\n",v_distal,v_proximal,v,I_proximal,I);}

  //Izhikevich model
  v=v+tau*(k*(v-vr)*(v-vt)-u+I)/C;
  u=u+tau*a*(b*(v-vr)-u);
  spike[number]=0;
  if(v>v_peak)
  {
    v=c;
    u=u+d;
    spike[number]=1;
  }


  neuro[number].v=v;
  neuro[number].u=u;
  Ix[number].I=0;
}

__global__ static void RS_neuron(int *input,struct axon *neuro,unsigned char *spike,struct neuron_I *Ix, int *boxnum, int *THREAD_NUM, int *BLOCK_NUM)
{
const int tid = threadIdx.x;
const int bid = blockIdx.x;
int number=(bid * THREAD_NUM[0] + tid)*10;


/********第一个神经元虚拟计算内核*********/
if((number+0)<=boxnum[0])
{RS(input,neuro,spike,Ix,number+0);}

/********第二个神经元虚拟计算内核********/
if((number+1)<=boxnum[0])
{RS(input,neuro,spike,Ix,number+1);}

/********第三个神经元虚拟计算内核********/
if((number+2)<=boxnum[0])
{RS(input,neuro,spike,Ix,number+2);}

/********第四个神经元虚拟计算内核*********/
if((number+3)<=boxnum[0])
{RS(input,neuro,spike,Ix,number+3);}

/********第五个神经元虚拟计算内核*********/
if((number+4)<=boxnum[0])
{RS(input,neuro,spike,Ix,number+4);}

/********第六个神经元虚拟计算内核*********/
if((number+5)<=boxnum[0])
{RS(input,neuro,spike,Ix,number+5);}

/********第七个神经元虚拟计算内核********/
if((number+6)<=boxnum[0])
{RS(input,neuro,spike,Ix,number+6);}

/********第八个神经元虚拟计算内核*********/
if((number+7)<=boxnum[0])
{RS(input,neuro,spike,Ix,number+7);}

/********第九个神经元虚拟计算内核*********/
if((number+8)<=boxnum[0])
{RS(input,neuro,spike,Ix,number+8);}

/********第十个神经元虚拟计算内核*********/
if((number+9)<=boxnum[0])
{RS(input,neuro,spike,Ix,number+9);}


}
