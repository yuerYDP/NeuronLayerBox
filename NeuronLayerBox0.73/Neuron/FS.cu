#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

__device__ static void FS(int *input,struct axon *neuro, unsigned char *spike,struct neuron_I *Ix,int number)
{
  //设置神经元计算参数
  float C=20;
  float k=1;
  float vr=-55;
  float vt=-40;
  float G_up=0.5;
  float G_down=1.0;
  float a=0.15;
  float b=8;
  float c=-55;
  float d=200;
  float v_peak=25;
  float I;

  float v=neuro[number].v;
  float u=neuro[number].u;
  I=Ix[number].I;


  //Izhikevich model
     v=v+tau*(k*(v-vr)*(v-vt)-u+I)/C;
     u=u+tau*a*(b*(v-vr)-u);
     spike[number]=0;
     if(v>v_peak)
     {
       v=c;
       u=u+d;
       spike[number]=1;
     }

  neuro[number].v=v;
  neuro[number].u=u;
  Ix[number].I=0;

}


__global__ static void FS_neuron(int *input,struct axon *neuro, unsigned char *spike,struct neuron_I *Ix, int *boxnum, int *THREAD_NUM, int *BLOCK_NUM)
{
const int tid = threadIdx.x;
const int bid = blockIdx.x;
int number=THREAD_NUM[0]*BLOCK_NUM[0]*10+(bid * THREAD_NUM[1] + tid)*10;


/********第一个神经元虚拟计算内核*********/
if((number+0)<=boxnum[1])
{FS(input,neuro,spike,Ix,number+0);}

/********第二个神经元虚拟计算内核********/
if((number+1)<=boxnum[1])
{FS(input,neuro,spike,Ix,number+1);}

/********第三个神经元虚拟计算内核********/
if((number+2)<=boxnum[1])
{FS(input,neuro,spike,Ix,number+2);}

/********第四个神经元虚拟计算内核*********/
if((number+3)<=boxnum[1])
{FS(input,neuro,spike,Ix,number+3);}

/********第五个神经元虚拟计算内核*********/
if((number+4)<=boxnum[1])
{FS(input,neuro,spike,Ix,number+4);}

/********第六个神经元虚拟计算内核*********/
if((number+5)<=boxnum[1])
{FS(input,neuro,spike,Ix,number+5);}

/********第七个神经元虚拟计算内核********/
if((number+6)<=boxnum[1])
{FS(input,neuro,spike,Ix,number+6);}

/********第八个神经元虚拟计算内核*********/
if((number+7)<=boxnum[1])
{FS(input,neuro,spike,Ix,number+7);}

/********第九个神经元虚拟计算内核*********/
if((number+8)<=boxnum[1])
{FS(input,neuro,spike,Ix,number+8);}

/********第十个神经元虚拟计算内核*********/
if((number+9)<=boxnum[1])
{FS(input,neuro,spike,Ix,number+9);}


}
