#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ static void post_Synapse(struct  synapse *syn, struct axon *neuro,unsigned char *spike,int *Ix,int *THREAD_SYN,int *BLOCK_SYN)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  int number=(bid * THREAD_SYN[1] + tid)*10;

  float w;
	float timeN;
  float v;
	float gsyn;
  float I_AMPA;
  float I_NMDA;
  float f;

/*************************突触第一颗虚拟计算单元******************************/
	w=syn[number].w;
	timeN=syn[number].timeN;
  v=neuro[syn[number].postsynapse_number].v;
  timeN++;

  if(spike[syn[number].persynapse_number]==1)//判断突触前神经元是否有信号
  {
  	timeN=0;
  }

  gsyn=w*gmax*A*(expf(-(timeN*tau)/t_1)-expf(-(timeN*tau)/t_2))/(t_1-t_2);
  if(timeN>1000000)
	{
	  timeN=1000000;
	  gsyn=0;
	}
  I_AMPA=gsyn*(Esyn-v);
  I_NMDA=(d1/(d1+d2*d3*expf(d4*v)))*gsyn*(Esyn-v);   //  I_NMDA=(1.0/(1+0.33*2*expf(-0.11*v)))*gsyn*(v-Esyn);
//  if(I_NMDA<0){I_NMDA=0.000001;}
//  if(I_AMPA<0){I_AMPA=0.000001;}

  atomicAdd(&Ix[syn[number].postsynapse_number],int(I_AMPA+I_NMDA)); //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_AMPA+I_NMDA;  //ԭ�Ӳ���

  if(spike[syn[number].postsynapse_number]==1)//判断突触后神经元是否有信号
  {
  	f=0.5;
  }
  else if(timeN<=1)
  {
  	f=-1.5;
  }
  else if(timeN<=126)
  {
  	f=(timeN-1)*0.2*0.02-0.5;
  }
  else
  {
    f=0;
  }
  w=w+tau*v*v*(gsyn*f*(-2)+1-w);

  syn[number].w=w;
  syn[number].timeN=timeN;
/************************************************************************/

/*************************突触第二颗虚拟计算单元******************************/
	w=syn[number+1].w;
	timeN=syn[number+1].timeN;
  v=neuro[syn[number+1].postsynapse_number].v;
  timeN++;

  if(spike[syn[number+1].persynapse_number]==1)//判断突触前神经元是否有信号
  {
  	timeN=0;
  }

  gsyn=w*gmax*A*(expf(-(timeN*tau)/t_1)-expf(-(timeN*tau)/t_2))/(t_1-t_2);
  if(timeN>1000000)
	{
	  timeN=1000000;
	  gsyn=0;
	}
  I_AMPA=gsyn*(Esyn-v);
  I_NMDA=(d1/(d1+d2*d3*expf(d4*v)))*gsyn*(Esyn-v);   //  I_NMDA=(1.0/(1+0.33*2*expf(-0.11*v)))*gsyn*(v-Esyn);
//  if(I_NMDA<0){I_NMDA=0.000001;}
//  if(I_AMPA<0){I_AMPA=0.000001;}

  atomicAdd(&Ix[syn[number+1].postsynapse_number],int(I_AMPA+I_NMDA)); //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_AMPA+I_NMDA;  //ԭ�Ӳ���

  if(spike[syn[number+1].postsynapse_number]==1)//判断突触后神经元是否有信号
  {
  	f=0.5;
  }
  else if(timeN<=1)
  {
  	f=-1.5;
  }
  else if(timeN<=126)
  {
  	f=(timeN-1)*0.2*0.02-0.5;
  }
  else
  {
    f=0;
  }
  w=w+tau*v*v*(gsyn*f*(-2)+1-w);

  syn[number].w=w;
  syn[number].timeN=timeN;
/************************************************************************/

/*************************突触第三颗虚拟计算单元******************************/
	w=syn[number+2].w;
	timeN=syn[number+2].timeN;
  v=neuro[syn[number+2].postsynapse_number].v;
  timeN++;

  if(spike[syn[number+2].persynapse_number]==1)//判断突触前神经元是否有信号
  {
  	timeN=0;
  }

  gsyn=w*gmax*A*(expf(-(timeN*tau)/t_1)-expf(-(timeN*tau)/t_2))/(t_1-t_2);
  if(timeN>1000000)
	{
	  timeN=1000000;
	  gsyn=0;
	}
  I_AMPA=gsyn*(Esyn-v);
  I_NMDA=(d1/(d1+d2*d3*expf(d4*v)))*gsyn*(Esyn-v);   //  I_NMDA=(1.0/(1+0.33*2*expf(-0.11*v)))*gsyn*(v-Esyn);
//  if(I_NMDA<0){I_NMDA=0.000001;}
//  if(I_AMPA<0){I_AMPA=0.000001;}

  atomicAdd(&Ix[syn[number+2].postsynapse_number],int(I_AMPA+I_NMDA)); //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_AMPA+I_NMDA;  //ԭ�Ӳ���

  if(spike[syn[number+2].postsynapse_number]==1)//判断突触后神经元是否有信号
  {
  	f=0.5;
  }
  else if(timeN<=1)
  {
  	f=-1.5;
  }
  else if(timeN<=126)
  {
  	f=(timeN-1)*0.2*0.02-0.5;
  }
  else
  {
    f=0;
  }
  w=w+tau*v*v*(gsyn*f*(-2)+1-w);

  syn[number+2].w=w;
  syn[number+2].timeN=timeN;
/************************************************************************/

/*************************突触第四颗虚拟计算单元******************************/
	w=syn[number+3].w;
	timeN=syn[number+3].timeN;
  v=neuro[syn[number+3].postsynapse_number].v;
  timeN++;

  if(spike[syn[number+3].persynapse_number]==1)//判断突触前神经元是否有信号
  {
  	timeN=0;
  }

  gsyn=w*gmax*A*(expf(-(timeN*tau)/t_1)-expf(-(timeN*tau)/t_2))/(t_1-t_2);
  if(timeN>1000000)
	{
	  timeN=1000000;
	  gsyn=0;
	}
  I_AMPA=gsyn*(Esyn-v);
  I_NMDA=(d1/(d1+d2*d3*expf(d4*v)))*gsyn*(Esyn-v);   //  I_NMDA=(1.0/(1+0.33*2*expf(-0.11*v)))*gsyn*(v-Esyn);
//  if(I_NMDA<0){I_NMDA=0.000001;}
//  if(I_AMPA<0){I_AMPA=0.000001;}

  atomicAdd(&Ix[syn[number+3].postsynapse_number],int(I_AMPA+I_NMDA)); //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_AMPA+I_NMDA;  //ԭ�Ӳ���

  if(spike[syn[number+3].postsynapse_number]==1)//判断突触后神经元是否有信号
  {
  	f=0.5;
  }
  else if(timeN<=1)
  {
  	f=-1.5;
  }
  else if(timeN<=126)
  {
  	f=(timeN-1)*0.2*0.02-0.5;
  }
  else
  {
    f=0;
  }
  w=w+tau*v*v*(gsyn*f*(-2)+1-w);

  syn[number+3].w=w;
  syn[number+3].timeN=timeN;
/************************************************************************/

/*************************突触第五颗虚拟计算单元******************************/
	w=syn[number+4].w;
	timeN=syn[number+4].timeN;
  v=neuro[syn[number+4].postsynapse_number].v;
  timeN++;

  if(spike[syn[number+4].persynapse_number]==1)//判断突触前神经元是否有信号
  {
  	timeN=0;
  }

  gsyn=w*gmax*A*(expf(-(timeN*tau)/t_1)-expf(-(timeN*tau)/t_2))/(t_1-t_2);
  if(timeN>1000000)
	{
	  timeN=1000000;
	  gsyn=0;
	}
  I_AMPA=gsyn*(Esyn-v);
  I_NMDA=(d1/(d1+d2*d3*expf(d4*v)))*gsyn*(Esyn-v);   //  I_NMDA=(1.0/(1+0.33*2*expf(-0.11*v)))*gsyn*(v-Esyn);
//  if(I_NMDA<0){I_NMDA=0.000001;}
//  if(I_AMPA<0){I_AMPA=0.000001;}

  atomicAdd(&Ix[syn[number+4].postsynapse_number],int(I_AMPA+I_NMDA)); //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_AMPA+I_NMDA;  //ԭ�Ӳ���

  if(spike[syn[number+4].postsynapse_number]==1)//判断突触后神经元是否有信号
  {
  	f=0.5;
  }
  else if(timeN<=1)
  {
  	f=-1.5;
  }
  else if(timeN<=126)
  {
  	f=(timeN-1)*0.2*0.02-0.5;
  }
  else
  {
    f=0;
  }
  w=w+tau*v*v*(gsyn*f*(-2)+1-w);

  syn[number+4].w=w;
  syn[number+4].timeN=timeN;
/************************************************************************/

/*************************突触第六颗虚拟计算单元******************************/
	w=syn[number+5].w;
	timeN=syn[number+5].timeN;
  v=neuro[syn[number+5].postsynapse_number].v;
  timeN++;

  if(spike[syn[number+5].persynapse_number]==1)//判断突触前神经元是否有信号
  {
  	timeN=0;
  }

  gsyn=w*gmax*A*(expf(-(timeN*tau)/t_1)-expf(-(timeN*tau)/t_2))/(t_1-t_2);
  if(timeN>1000000)
	{
	  timeN=1000000;
	  gsyn=0;
	}
  I_AMPA=gsyn*(Esyn-v);
  I_NMDA=(d1/(d1+d2*d3*expf(d4*v)))*gsyn*(Esyn-v);   //  I_NMDA=(1.0/(1+0.33*2*expf(-0.11*v)))*gsyn*(v-Esyn);
//  if(I_NMDA<0){I_NMDA=0.000001;}
//  if(I_AMPA<0){I_AMPA=0.000001;}

  atomicAdd(&Ix[syn[number+5].postsynapse_number],int(I_AMPA+I_NMDA)); //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_AMPA+I_NMDA;  //ԭ�Ӳ���

  if(spike[syn[number+5].postsynapse_number]==1)//判断突触后神经元是否有信号
  {
  	f=0.5;
  }
  else if(timeN<=1)
  {
  	f=-1.5;
  }
  else if(timeN<=126)
  {
  	f=(timeN-1)*0.2*0.02-0.5;
  }
  else
  {
    f=0;
  }
  w=w+tau*v*v*(gsyn*f*(-2)+1-w);

  syn[number+5].w=w;
  syn[number+5].timeN=timeN;
/************************************************************************/

/*************************突触第七颗虚拟计算单元******************************/
	w=syn[number+6].w;
	timeN=syn[number+6].timeN;
  v=neuro[syn[number+6].postsynapse_number].v;
  timeN++;

  if(spike[syn[number+6].persynapse_number]==1)//判断突触前神经元是否有信号
  {
  	timeN=0;
  }

  gsyn=w*gmax*A*(expf(-(timeN*tau)/t_1)-expf(-(timeN*tau)/t_2))/(t_1-t_2);
  if(timeN>1000000)
	{
	  timeN=1000000;
	  gsyn=0;
	}
  I_AMPA=gsyn*(Esyn-v);
  I_NMDA=(d1/(d1+d2*d3*expf(d4*v)))*gsyn*(Esyn-v);   //  I_NMDA=(1.0/(1+0.33*2*expf(-0.11*v)))*gsyn*(v-Esyn);
//  if(I_NMDA<0){I_NMDA=0.000001;}
//  if(I_AMPA<0){I_AMPA=0.000001;}

  atomicAdd(&Ix[syn[number+6].postsynapse_number],int(I_AMPA+I_NMDA)); //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_AMPA+I_NMDA;  //ԭ�Ӳ���

  if(spike[syn[number+6].postsynapse_number]==1)//判断突触后神经元是否有信号
  {
  	f=0.5;
  }
  else if(timeN<=1)
  {
  	f=-1.5;
  }
  else if(timeN<=126)
  {
  	f=(timeN-1)*0.2*0.02-0.5;
  }
  else
  {
    f=0;
  }
  w=w+tau*v*v*(gsyn*f*(-2)+1-w);

  syn[number+6].w=w;
  syn[number+6].timeN=timeN;
/************************************************************************/

/*************************突触第八颗虚拟计算单元******************************/
	w=syn[number+7].w;
	timeN=syn[number+7].timeN;
  v=neuro[syn[number+7].postsynapse_number].v;
  timeN++;

  if(spike[syn[number+7].persynapse_number]==1)//判断突触前神经元是否有信号
  {
  	timeN=0;
  }

  gsyn=w*gmax*A*(expf(-(timeN*tau)/t_1)-expf(-(timeN*tau)/t_2))/(t_1-t_2);
  if(timeN>1000000)
	{
	  timeN=1000000;
	  gsyn=0;
	}
  I_AMPA=gsyn*(Esyn-v);
  I_NMDA=(d1/(d1+d2*d3*expf(d4*v)))*gsyn*(Esyn-v);   //  I_NMDA=(1.0/(1+0.33*2*expf(-0.11*v)))*gsyn*(v-Esyn);
//  if(I_NMDA<0){I_NMDA=0.000001;}
//  if(I_AMPA<0){I_AMPA=0.000001;}

  atomicAdd(&Ix[syn[number+7].postsynapse_number],int(I_AMPA+I_NMDA)); //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_AMPA+I_NMDA;  //ԭ�Ӳ���

  if(spike[syn[number+7].postsynapse_number]==1)//判断突触后神经元是否有信号
  {
  	f=0.5;
  }
  else if(timeN<=1)
  {
  	f=-1.5;
  }
  else if(timeN<=126)
  {
  	f=(timeN-1)*0.2*0.02-0.5;
  }
  else
  {
    f=0;
  }
  w=w+tau*v*v*(gsyn*f*(-2)+1-w);

  syn[number+7].w=w;
  syn[number+7].timeN=timeN;
/************************************************************************/

/*************************突触第九颗虚拟计算单元******************************/
	w=syn[number+8].w;
	timeN=syn[number+8].timeN;
  v=neuro[syn[number+8].postsynapse_number].v;
  timeN++;

  if(spike[syn[number+8].persynapse_number]==1)//判断突触前神经元是否有信号
  {
  	timeN=0;
  }

  gsyn=w*gmax*A*(expf(-(timeN*tau)/t_1)-expf(-(timeN*tau)/t_2))/(t_1-t_2);
  if(timeN>1000000)
	{
	  timeN=1000000;
	  gsyn=0;
	}
  I_AMPA=gsyn*(Esyn-v);
  I_NMDA=(d1/(d1+d2*d3*expf(d4*v)))*gsyn*(Esyn-v);   //  I_NMDA=(1.0/(1+0.33*2*expf(-0.11*v)))*gsyn*(v-Esyn);
//  if(I_NMDA<0){I_NMDA=0.000001;}
//  if(I_AMPA<0){I_AMPA=0.000001;}

  atomicAdd(&Ix[syn[number+8].postsynapse_number],int(I_AMPA+I_NMDA)); //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_AMPA+I_NMDA;  //ԭ�Ӳ���

  if(spike[syn[number+8].postsynapse_number]==1)//判断突触后神经元是否有信号
  {
  	f=0.5;
  }
  else if(timeN<=1)
  {
  	f=-1.5;
  }
  else if(timeN<=126)
  {
  	f=(timeN-1)*0.2*0.02-0.5;
  }
  else
  {
    f=0;
  }
  w=w+tau*v*v*(gsyn*f*(-2)+1-w);

  syn[number+8].w=w;
  syn[number+8].timeN=timeN;
/************************************************************************/

/*************************突触第十颗虚拟计算单元******************************/
	w=syn[number+9].w;
	timeN=syn[number+9].timeN;
  v=neuro[syn[number+9].postsynapse_number].v;
  timeN++;

  if(spike[syn[number+9].persynapse_number]==1)//判断突触前神经元是否有信号
  {
  	timeN=0;
  }

  gsyn=w*gmax*A*(expf(-(timeN*tau)/t_1)-expf(-(timeN*tau)/t_2))/(t_1-t_2);
  if(timeN>1000000)
	{
	  timeN=1000000;
	  gsyn=0;
	}
  I_AMPA=gsyn*(Esyn-v);
  I_NMDA=(d1/(d1+d2*d3*expf(d4*v)))*gsyn*(Esyn-v);   //  I_NMDA=(1.0/(1+0.33*2*expf(-0.11*v)))*gsyn*(v-Esyn);
//  if(I_NMDA<0){I_NMDA=0.000001;}
//  if(I_AMPA<0){I_AMPA=0.000001;}

  atomicAdd(&Ix[syn[number+9].postsynapse_number],int(I_AMPA+I_NMDA)); //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_AMPA+I_NMDA;  //ԭ�Ӳ���

  if(spike[syn[number+9].postsynapse_number]==1)//判断突触后神经元是否有信号
  {
  	f=0.5;
  }
  else if(timeN<=1)
  {
  	f=-1.5;
  }
  else if(timeN<=126)
  {
  	f=(timeN-1)*0.2*0.02-0.5;
  }
  else
  {
    f=0;
  }
  w=w+tau*v*v*(gsyn*f*(-2)+1-w);

  syn[number+9].w=w;
  syn[number+9].timeN=timeN;
/************************************************************************/
}
