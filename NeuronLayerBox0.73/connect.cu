#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <signal.h>
#include <sys/time.h>

struct syn_loaddata
{
  int *type;
  int *Prelayer;
  int *Postlayer;
  int *inter;
  int *outer;
  int *connect_point;
  float *g1;
  float *g2;
  float *t1r;
  float *t1f;
  float *t2r;
  float *t2f;
  float *axon_delay;
};

int compute_num(int *X,int *Y,struct syn_loaddata *syn_data,int n);

/********************************************analysis***************************************/
int analysis_synapse_func(struct syn_loaddata *syn_data,int *X_max,int *Y_max,int maxline,int box,int *_THREAD_SYN,int *_BLOCK_SYN)
{
  int i;
  int num=0;
  for(i=0;i<maxline;i++)
  {
    if(syn_data->type[i]==box)
    {
        if(X_max[syn_data->Prelayer[i]]<=0 || Y_max[syn_data->Prelayer[i]]<=0 || X_max[syn_data->Postlayer[i]]<=0 || Y_max[syn_data->Postlayer[i]]<=0)
        {
          printf("error:xmax and ymax must >0\n");
          return -1;
        }
        num+=compute_num(X_max,Y_max,syn_data,i);
    }
  }
  if(num==0)
  {
    *_THREAD_SYN=0;
    *_BLOCK_SYN=0;
  }
  else if(num%1280==0)
  {
    *_BLOCK_SYN=num/1280;
    *_THREAD_SYN=128;
  }
  else if(num%960==0)
  {
    *_BLOCK_SYN=num/960;
    *_THREAD_SYN=96;
  }
  else if(num%640==0)
  {
    *_BLOCK_SYN=num/640;
    *_THREAD_SYN=64;
  }
  else if(num%320==0)
  {
    *_BLOCK_SYN=num/320;
    *_THREAD_SYN=32;
  }
  else
  {
    *_BLOCK_SYN=ceil(num/320.0);
    *_THREAD_SYN=32;
  }
  printf("num=%d\n",num);
  return 0;
}

int compute_num(int *X,int *Y,struct syn_loaddata *syn_data,int n)
{
  int all=0;
  FILE *fp=fopen("load_data/data.txt","ab");
  int x1,x2,y1,y2;
  x1=X[syn_data->Prelayer[n]];   //persynapse xmax
  y1=Y[syn_data->Prelayer[n]];   //persynapse ymax
  x2=X[syn_data->Postlayer[n]];   //postsynapse xmax
  y2=Y[syn_data->Postlayer[n]];   //postsynapse ymax
  float length1=syn_data->inter[n];
  float length2=syn_data->outer[n];
  for(int x_pre=0;x_pre<x1;x_pre++)
  {
    for(int y_pre=0;y_pre<y1;y_pre++)
    {
      int x_on=(int)(x_pre*x2*1.0/x1+0.5);
      int y_on=(int)(y_pre*y2*1.0/y1+0.5);
      for(int l_x=-length2;l_x<=length2;l_x++)
      {
        for(int l_y=-length2;l_y<=length2;l_y++)
        {
          float l=sqrt(l_x*l_x+l_y*l_y);
          if(l<=length2 && l>=length1)
          {
            if((x_on+l_x)>=0 && (x_on+l_x)<x2 && (y_on+l_y)>=0 && (y_on+l_y)<y2)
            {
              all+=1;
              int dataPtr[8];
              dataPtr[0]=syn_data->type[n];
              dataPtr[1]=n;
              dataPtr[2]=syn_data->Prelayer[n];
              dataPtr[3]=syn_data->Postlayer[n];
              dataPtr[4]=x_pre;
              dataPtr[5]=y_pre;
              dataPtr[6]=x_on+l_x;
              dataPtr[7]=y_on+l_y;
              for(int p=0;p<8;p++)
              {fprintf(fp, "%d ", dataPtr[p]);}
              fprintf(fp, "%f", l);
              fprintf(fp, "\n");
            }
          }
        }
      }
    }
  }
  fclose(fp);
  return all;
}


/**********************************connetc*************************************/
int connect_error(struct axon *neuron_copy,int *max,int pre_addr,int post_addr,int j,int k,int aj,int ak,int Prelayer,int Postlayer,int numbers)
{
  if(neuron_copy[pre_addr+j*max[Prelayer]+k].layer!=Prelayer || neuron_copy[post_addr+aj*max[Postlayer]+ak].layer!=Postlayer)
  {
    printf("synapse_cpoy_numbers=%d\n",numbers);
    printf("per=%d\n",pre_addr+(j)*max[Prelayer]+k);
    printf("post=%d\n",post_addr+(aj)*max[Postlayer]+ak);
    printf("per_layer=%d\n",neuron_copy[pre_addr+(j)*max[Prelayer]+k].layer);
    printf("post_layer=%d\n",neuron_copy[post_addr+(aj)*max[Postlayer]+ak].layer);
    printf("Prelayer=%d,Postlayer=%d\n",Prelayer,Postlayer);
    return -1;
  }
  return 0;
}

int addr_compute(struct axon *neuron_copy,int layer)
{
  int j=0;
  int layer_addr=-1;
  while(1)
  {
    if(neuron_copy[j].layer==layer)             //寻找突触前神经元对应neuronbox首地址
    {layer_addr=j;j=0;break;}
    j++;
  }
  return layer_addr;
}

int connect_synapse_func(struct syn_loaddata *syn_data,int *X_max,int *Y_max,int box,int *_THREAD_SYN,int *_BLOCK_SYN,struct synapse *synapse_cpoy,struct axon *neuron_copy,int *N_NUM)
{
  //syn_type:突触box类型
  //Prelayer:突触前层
  //Postlayer:突触后层
  //connect_point:突触连接点,[0,1,2]
  //X_max:各层神经元x最大尺寸
  //Y_max:各层神经元y最大尺寸
  //maxline:synapse.txt中总共层间连接数,文件中的每行相当于一个连接
  //box:突触的指定box类型(1,2,3,4,5,6)
  //synapse_cpoy:突触计算数据
  //neuron_copy:神经元计算数据
  //_THREAD_SYN,_BLOCK_SYN:cuda计算突触线程数相关


  int i;
  int j,k;
  int numbers=0;
  int all_num=0;
  int per_addr=0;
  int post_addr=0;

  int error=0;
  FILE *fp=fopen("load_data/data.txt","r");
  int m;
  int type,synlayer,Perlayer,Postlayer,x_per,y_per,x_post,y_post;
  int old_Perlayer=-1,old_Postlayer=-1;
  float l;
  int first_flag=0;
  while (!feof(fp))
  {
    fscanf(fp, "%d", &m);
    type=m;
    fscanf(fp, "%d", &m);
    synlayer=m;
    fscanf(fp, "%d", &m);
    Perlayer=m;
    fscanf(fp, "%d", &m);
    Postlayer=m;
    fscanf(fp, "%d", &m);
    x_per=m;
    fscanf(fp, "%d", &m);
    y_per=m;
    fscanf(fp, "%d", &m);
    x_post=m;
    fscanf(fp, "%d", &m);
    y_post=m;
    fscanf(fp, "%f", &l);
    all_num++;
    if(type==box)
    {
      if(first_flag==0){first_flag=1;}  //判断是否第一次进入相应的synapse box

      if(Perlayer!=old_Perlayer)
      {
        //printf("type=%d,synlayer=%d,perlayer=%d,postlayer=%d,x_per=%d,y_per=%d,x_post=%d,y_post=%d,l=%f\n",type,synlayer,Perlayer,Postlayer,x_per,y_per,x_post,y_post,l);
        printf("start per addr search\n");
        per_addr=addr_compute(neuron_copy,Perlayer);
        printf("per_addr=%d , Perlayer=%d\n",per_addr,Perlayer);
        printf("search per addr over\n");
        old_Perlayer=Perlayer;
      }
      if(Postlayer!=old_Postlayer)
      {
        printf("start post addr search\n");
        post_addr=addr_compute(neuron_copy,Postlayer);
        printf("post_addr=%d, Postlayer=%d\n",post_addr,Postlayer);
        printf("search post addr over\n");
        old_Postlayer=Postlayer;
      }


      //connect_point:突触连接点 0:神经元上 1:神经元树突附近  2:树突顶端
      //g1:NMDA or GABAa w*gpeak
      //g2:AMPA or GABAb w*gpeak
      //t1r:NMDA or GABAa计算参数
      //t1f:NMDA or GABAa计算参数
      //t2r:AMPA or GABAb计算参数
      //t2f:AMPA or GABAb计算参数
      //axon_delay:轴突延迟 tau的整数倍
      float w_ij;
      if(numbers>=(*_THREAD_SYN) *(*_BLOCK_SYN)*10){printf("error:number=%d,THREAD_SYN=%d,BLOCK_SYN=%d,box=%d\n",numbers,*_THREAD_SYN,*_BLOCK_SYN,box);error=-1;break;}
      if(syn_data->outer[synapse_cpoy[numbers].synaplayer]==0){w_ij=1;}
      else{w_ij=exp(-pow(l/syn_data->outer[synapse_cpoy[numbers].synaplayer],2));}
      synapse_cpoy[numbers].synaplayer=synlayer;
      synapse_cpoy[numbers].persynapse_number=per_addr+y_per*X_max[Perlayer]+x_per;
      synapse_cpoy[numbers].postsynapse_number=post_addr+y_post*X_max[Postlayer]+x_post;
      if(connect_error(neuron_copy,X_max,per_addr,post_addr,y_per,x_per,y_post,x_post,Perlayer,Postlayer,numbers)!=0){printf("connect_error\n");error=-1;break;}
      synapse_cpoy[numbers].w=w_init;
      synapse_cpoy[numbers].z=z_init;
      synapse_cpoy[numbers].timeN=0;
      synapse_cpoy[numbers].connect_point=syn_data->connect_point[synapse_cpoy[numbers].synaplayer];
      synapse_cpoy[numbers].g1=syn_data->g1[synapse_cpoy[numbers].synaplayer]*w_ij;
      synapse_cpoy[numbers].g2=syn_data->g2[synapse_cpoy[numbers].synaplayer]*w_ij;
      synapse_cpoy[numbers].s1=0;
      synapse_cpoy[numbers].R1=0;
      synapse_cpoy[numbers].s2=0;
      synapse_cpoy[numbers].R2=0;
      synapse_cpoy[numbers].t1_r=syn_data->t1r[synapse_cpoy[numbers].synaplayer];
      synapse_cpoy[numbers].t1_f=syn_data->t1f[synapse_cpoy[numbers].synaplayer];
      synapse_cpoy[numbers].t2_r=syn_data->t2r[synapse_cpoy[numbers].synaplayer];
      synapse_cpoy[numbers].t2_f=syn_data->t2f[synapse_cpoy[numbers].synaplayer];
      synapse_cpoy[numbers].axon_delay=floor(syn_data->axon_delay[synapse_cpoy[numbers].synaplayer]/tau_cpu);
      numbers++;
    }
    else
    {
      if(first_flag==1){break;}//已经进入相应的synapse box中途断开再出来后面没有相应的box，相同box是连续的。
    }
  }
  if(error<0){return -1;}
  N_NUM[box-1]=numbers-1;
  fclose(fp);
  printf("all_search=%d,number=%d\n",all_num,numbers);
  return 0;
}
