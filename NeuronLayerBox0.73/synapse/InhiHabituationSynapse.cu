#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

__device__ static void inhi_habi_syna(struct synapse *neuro_syn, struct axon *neuro, unsigned char *spike, struct neuron_I *Ix, int number)
{
  float ga;
  float gb;
  float t_f;
  float t_r;
  float gpeak;
  float timeN;
  float v;
  int I;
  float p;             //max g(t)
  float dt;
  float z;
  float z_spike;
  unsigned long int prespike;
  float T,s,R;
  float t_max,g2_peak;

  timeN=neuro_syn[number].timeN;
  z=neuro_syn[number].z;
  v=neuro[neuro_syn[number].postsynapse_number].v;
  timeN++;
  if(timeN>1000000)
  {
   timeN=1000000;
  }

  //habituation
  z=z+tau*(1-z)/400;
  //GABAa
  t_f=neuro_syn[number].t1_f;
  t_r=neuro_syn[number].t1_r;
  gpeak=neuro_syn[number].g1;
  s=neuro_syn[number].s1;
  if(timeN*tau<t_r){T=1.0/t_r;}
  else{T=0;}
  if(t_r==t_f)
  {
    s=s+tau*(T*(1-s)-s/t_f);
    ga=gpeak*s*g1_peak;
  }
  else
  {
    R=neuro_syn[number].R1;
    t_max=(t_f*t_r/(t_f-t_r))*log(t_f/t_r);
    g2_peak=1+(t_r/t_f)/(2*R_max*expf((t_r-t_max)/t_r));
    R=R+tau*((1.0-R)*T-R/t_r);
    s=s+tau*((t_f+t_r)/t_f)*((2.0/t_r)*(1.0-s)*R-(s/t_f));
    ga=gpeak*s*g2_peak;
  }
  neuro_syn[number].s1=s;
  neuro_syn[number].R1=R;

  //GABAb
  t_f=neuro_syn[number].t2_f;
  t_r=neuro_syn[number].t2_r;
  gpeak=neuro_syn[number].g2;
  s=neuro_syn[number].s2;
  if(timeN*tau<t_r){T=1.0/t_r;}
  else{T=0;}
  if(t_r==t_f)
  {
    s=s+tau*(T*(1-s)-s/t_f);
    gb=gpeak*s*g1_peak;
  }
  else
  {
    R=neuro_syn[number].R2;
    t_max=(t_f*t_r/(t_f-t_r))*log(t_f/t_r);
    g2_peak=1+(t_r/t_f)/(2*R_max*expf((t_r-t_max)/t_r));
    R=R+tau*((1.0-R)*T-R/t_r);
    s=s+tau*((t_f+t_r)/t_f)*((2.0/t_r)*(1.0-s)*R-(s/t_f));
    gb=gpeak*s*g2_peak;
  }
  neuro_syn[number].s2=s;
  neuro_syn[number].R2=R;

  z_spike=neuro_syn[number].z_spike;
  I=z_spike*ga*(-70-v)+gb*(-90-v);
  atomicAdd(&Ix[neuro_syn[number].postsynapse_number].I,I);

  prespike=(neuro_syn[number].axon<<1) | spike[neuro_syn[number].persynapse_number];
  if(((prespike>>neuro_syn[number].axon_delay)&0x01)==1) //判断突触前神经元是否有信号
  {
    timeN=0;
    neuro_syn[number].z_spike=z;
    z=0;
  }
  neuro_syn[number].axon=prespike;

  neuro_syn[number].timeN=timeN;
  neuro_syn[number].z=z;

}

__global__ static void inhi_habituation_Synapse(struct synapse *neuro_syn, struct axon *neuro,unsigned char *spike,struct neuron_I *Ix,int *boxnum,int *THREAD_SYN,int *BLOCK_SYN)
{
   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   int number=(bid * THREAD_SYN[5] + tid)*10;


/*************************突触第一颗虚拟计算单元******************************/
if(number+0<=boxnum[5])
{inhi_habi_syna(neuro_syn,neuro,spike,Ix,number+0);}
/********************************************************************************/

/*************************突触第二颗虚拟计算单元******************************/
if(number+1<=boxnum[5])
{inhi_habi_syna(neuro_syn,neuro,spike,Ix,number+1);}
/********************************************************************************/

/*************************突触第三颗虚拟计算单元******************************/
if(number+2<=boxnum[5])
{inhi_habi_syna(neuro_syn,neuro,spike,Ix,number+2);}
/********************************************************************************/

/*************************突触第四颗虚拟计算单元******************************/
if(number+3<=boxnum[5])
{inhi_habi_syna(neuro_syn,neuro,spike,Ix,number+3);}
/********************************************************************************/

/*************************突触第五颗虚拟计算单元******************************/
if(number+4<=boxnum[5])
{inhi_habi_syna(neuro_syn,neuro,spike,Ix,number+4);}
/********************************************************************************/

/*************************突触第六颗虚拟计算单元******************************/
if(number+5<=boxnum[5])
{inhi_habi_syna(neuro_syn,neuro,spike,Ix,number+5);}
/********************************************************************************/

/*************************突触第七颗虚拟计算单元******************************/
if(number+6<=boxnum[5])
{inhi_habi_syna(neuro_syn,neuro,spike,Ix,number+6);}
/********************************************************************************/

/*************************突触第八颗虚拟计算单元******************************/
if(number+7<=boxnum[5])
{inhi_habi_syna(neuro_syn,neuro,spike,Ix,number+7);}
/********************************************************************************/

/*************************突触第九颗虚拟计算单元******************************/
if(number+8<=boxnum[5])
{inhi_habi_syna(neuro_syn,neuro,spike,Ix,number+8);}
/********************************************************************************/

/*************************突触第十颗虚拟计算单元******************************/
if(number+9<=boxnum[5])
{inhi_habi_syna(neuro_syn,neuro,spike,Ix,number+9);}
/********************************************************************************/


}
