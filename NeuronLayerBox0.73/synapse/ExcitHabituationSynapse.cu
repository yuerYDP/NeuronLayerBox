#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

__device__ static void ex_habi_syna(struct synapse *neuro_syn, struct axon *neuro,unsigned char *spike,struct neuron_I *Ix,int number)
{
  float g_NMDA;
  float g_AMPA;
  float t_f;
  float t_r;
  float gpeak;
  float timeN;
  float v;
  int I;
  float p;             //max g(t)
  float dt;
  float g_scale;
  float z;
  float z_spike;
  unsigned long int prespike;
  float T,s,R;
  float t_max,g2_peak;

  timeN=neuro_syn[number].timeN;
  z=neuro_syn[number].z;
  v=neuro[neuro_syn[number].postsynapse_number].v;
  timeN++;
  if(timeN>1000000)
  {
   timeN=1000000;
  }

  //habituation
  z=z+tau*(1-z)/400;
  //NMDA
  t_f=neuro_syn[number].t1_f;
  t_r=neuro_syn[number].t1_r;
  gpeak=neuro_syn[number].g1;
  g_scale=1/(1+expf((-v-25)/12.5));
  s=neuro_syn[number].s1;
  if(timeN*tau<t_r){T=1.0/t_r;}
  else{T=0;}
  if(t_r==t_f)
  {
    s=s+tau*(T*(1-s)-s/t_f);
    g_NMDA=g_scale*gpeak*s*g1_peak;
  }
  else
  {
    R=neuro_syn[number].R1;
    t_max=(t_f*t_r/(t_f-t_r))*log(t_f/t_r);
    g2_peak=1+(t_r/t_f)/(2*R_max*expf((t_r-t_max)/t_r));
    R=R+tau*((1.0-R)*T-R/t_r);
    s=s+tau*((t_f+t_r)/t_f)*((2.0/t_r)*(1.0-s)*R-(s/t_f));
    g_NMDA=g_scale*gpeak*s*g2_peak;
  }
  neuro_syn[number].s1=s;
  neuro_syn[number].R1=R;

  //AMPA
  t_f=neuro_syn[number].t2_f;
  t_r=neuro_syn[number].t2_r;
  gpeak=neuro_syn[number].g2;
  s=neuro_syn[number].s2;
  if(timeN*tau<t_r){T=1.0/t_r;}
  else{T=0;}
  if(t_r==t_f)
  {
    s=s+tau*(T*(1-s)-s/t_f);
    g_AMPA=gpeak*s*g1_peak;
  }
  else
  {
    R=neuro_syn[number].R2;
    t_max=(t_f*t_r/(t_f-t_r))*log(t_f/t_r);
    g2_peak=1+(t_r/t_f)/(2*R_max*expf((t_r-t_max)/t_r));
    R=R+tau*((1.0-R)*T-R/t_r);
    s=s+tau*((t_f+t_r)/t_f)*((2.0/t_r)*(1.0-s)*R-(s/t_f));
    g_AMPA=gpeak*s*g2_peak;
  }
  neuro_syn[number].s2=s;
  neuro_syn[number].R2=R;

  z_spike=neuro_syn[number].z_spike;
  I=g_NMDA*(0-v)+z_spike*g_AMPA*(0-v);
  atomicAdd(&Ix[neuro_syn[number].postsynapse_number].I,I); //  Ix[neuro_syn[number].postsynapse_number]+=I_AMPA+I_NMDA;

  //if(neuro[neuro_syn[number].persynapse_number].layer==5 && neuro[neuro_syn[number].postsynapse_number].layer==7 && neuro_syn[number].postsynapse_number==600000)
  //{printf("g_AMPA:%f  Ix:%d  timeN:%f  T:%f  g2_peak:%f  s=%f  v=%f  z_spike=%f\n",g_AMPA,Ix[neuro_syn[number].postsynapse_number].I,timeN,T,g2_peak,s,v,z_spike);}

  prespike=(neuro_syn[number].axon<<1) | spike[neuro_syn[number].persynapse_number];
  if(((prespike>>neuro_syn[number].axon_delay)&0x01)==1) //判断突触前神经元是否有信号
  {
    timeN=0;
    neuro_syn[number].z_spike=z;
    z=0;
  }
  neuro_syn[number].axon=prespike;

  neuro_syn[number].timeN=timeN;
  neuro_syn[number].z=z;
}

__global__ static void excit_habituation_Synapse(struct synapse *neuro_syn, struct axon *neuro,unsigned char *spike,struct neuron_I *Ix,int *boxnum,int *THREAD_SYN,int *BLOCK_SYN)
{
   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   int number=(bid * THREAD_SYN[4] + tid)*10;


/*************************突触第一颗虚拟计算单元******************************/
if(number+0<=boxnum[4])
{ex_habi_syna(neuro_syn, neuro, spike, Ix, number+0);}

/********************************************************************************/

/*************************突触第二颗虚拟计算单元******************************/
if(number+1<=boxnum[4])
{ex_habi_syna(neuro_syn, neuro, spike, Ix, number+1);}

/********************************************************************************/

/*************************突触第三颗虚拟计算单元******************************/
if(number+2<=boxnum[4])
{ex_habi_syna(neuro_syn, neuro, spike, Ix, number+2);}

/********************************************************************************/

/*************************突触第四颗虚拟计算单元******************************/
if(number+3<=boxnum[4])
{ex_habi_syna(neuro_syn, neuro, spike, Ix, number+3);}

/********************************************************************************/

/*************************突触第五颗虚拟计算单元******************************/
if(number+4<=boxnum[4])
{ex_habi_syna(neuro_syn, neuro, spike, Ix, number+4);}

/********************************************************************************/

/*************************突触第六颗虚拟计算单元******************************/
if(number+5<=boxnum[4])
{ex_habi_syna(neuro_syn, neuro, spike, Ix, number+5);}

/********************************************************************************/

/*************************突触第七颗虚拟计算单元******************************/
if(number+6<=boxnum[4])
{ex_habi_syna(neuro_syn, neuro, spike, Ix, number+6);}

/********************************************************************************/

/*************************突触第八颗虚拟计算单元******************************/
if(number+7<=boxnum[4])
{ex_habi_syna(neuro_syn, neuro, spike, Ix, number+7);}

/********************************************************************************/

/*************************突触第九颗虚拟计算单元******************************/
if(number+8<=boxnum[4])
{ex_habi_syna(neuro_syn, neuro, spike, Ix, number+8);}

/********************************************************************************/

/*************************突触第十颗虚拟计算单元******************************/
if(number+9<=boxnum[4])
{ex_habi_syna(neuro_syn, neuro, spike, Ix, number+9);}

/********************************************************************************/

}
