#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

__device__ static void inhi_syna(struct synapse *neuro_syn, struct axon *neuro,unsigned char *spike,struct neuron_I *Ix,int number)
{
  float ga;
  float gb;
  float t_f;
  float t_r;
  float gpeak;
  float timeN;
  float v;
  int I;
  float p;             //max g(t)
  float dt;
  unsigned long int prespike;
  float T,s,R;
  float t_max,g2_peak;

  timeN=neuro_syn[number].timeN;
  v=neuro[neuro_syn[number].postsynapse_number].v;
  timeN++;
  if(timeN>1000000)
  {
   timeN=1000000;
  }

  //GABAa
  t_f=neuro_syn[number].t1_f;
  t_r=neuro_syn[number].t1_r;
  gpeak=neuro_syn[number].g1;
  s=neuro_syn[number].s1;
  if(timeN*tau<t_r){T=1.0/t_r;}
  else{T=0;}
  if(t_r==t_f)
  {
    s=s+tau*(T*(1-s)-s/t_f);
    ga=gpeak*s*g1_peak;
  }
  else
  {
    R=neuro_syn[number].R1;
    t_max=(t_f*t_r/(t_f-t_r))*log(t_f/t_r);
    g2_peak=1+(t_r/t_f)/(2*R_max*expf((t_r-t_max)/t_r));
    R=R+tau*((1.0-R)*T-R/t_r);
    s=s+tau*((t_f+t_r)/t_f)*((2.0/t_r)*(1.0-s)*R-(s/t_f));
    ga=gpeak*s*g2_peak;
  }
  neuro_syn[number].s1=s;
  neuro_syn[number].R1=R;

  //GABAb
  t_f=neuro_syn[number].t2_f;
  t_r=neuro_syn[number].t2_r;
  gpeak=neuro_syn[number].g2;
  s=neuro_syn[number].s2;
  if(timeN*tau<t_r){T=1.0/t_r;}
  else{T=0;}
  if(t_r==t_f)
  {
    s=s+tau*(T*(1-s)-s/t_f);
    gb=gpeak*s*g1_peak;
  }
  else
  {
    R=neuro_syn[number].R2;
    t_max=(t_f*t_r/(t_f-t_r))*log(t_f/t_r);
    g2_peak=1+(t_r/t_f)/(2*R_max*expf((t_r-t_max)/t_r));
    R=R+tau*((1.0-R)*T-R/t_r);
    s=s+tau*((t_f+t_r)/t_f)*((2.0/t_r)*(1.0-s)*R-(s/t_f));
    gb=gpeak*s*g2_peak;
  }
  neuro_syn[number].s2=s;
  neuro_syn[number].R2=R;


  I=ga*(-70-v)+gb*(-90-v);
  atomicAdd(&Ix[neuro_syn[number].postsynapse_number].I,I);  //  Ix[syn[bid * THREAD_NUM + tid].postsynapse_number]+=I_GABAa+I_GABAb;

  prespike=(neuro_syn[number].axon<<1) | spike[neuro_syn[number].persynapse_number];
  if(((prespike>>neuro_syn[number].axon_delay)&0x01)==1) //判断突触前神经元是否有信号
  {
    timeN=0;
    //if(neuro[neuro_syn[number].persynapse_number].layer==7 && neuro[neuro_syn[number].postsynapse_number].layer==8)
    //{printf("ga:%f  Ix:%d  gpeak:%f  T:%f\n",ga,Ix[neuro_syn[number].postsynapse_number].I,gpeak,T);}
  }
  neuro_syn[number].axon=prespike;

  neuro_syn[number].timeN=timeN;

}

__global__ static void inhi_Synapse(struct synapse *neuro_syn, struct axon *neuro,unsigned char *spike,struct neuron_I *Ix,int *boxnum,int *THREAD_SYN,int *BLOCK_SYN)
{
   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   int number=(bid * THREAD_SYN[3] + tid)*10;


/*************************突触第一颗虚拟计算单元******************************/
if(number+0<=boxnum[3])
{inhi_syna(neuro_syn,neuro,spike,Ix,number+0);}
/********************************************************************************/

/*************************突触第二颗虚拟计算单元******************************/
if(number+1<=boxnum[3])
{inhi_syna(neuro_syn,neuro,spike,Ix,number+1);}
/********************************************************************************/

/*************************突触第三颗虚拟计算单元******************************/
if(number+2<=boxnum[3])
{inhi_syna(neuro_syn,neuro,spike,Ix,number+2);}
/********************************************************************************/

/*************************突触第四颗虚拟计算单元******************************/
if(number+3<=boxnum[3])
{inhi_syna(neuro_syn,neuro,spike,Ix,number+3);}
/********************************************************************************/

/*************************突触第五颗虚拟计算单元******************************/
if(number+4<=boxnum[3])
{inhi_syna(neuro_syn,neuro,spike,Ix,number+4);}
/********************************************************************************/

/*************************突触第六颗虚拟计算单元******************************/
if(number+5<=boxnum[3])
{inhi_syna(neuro_syn,neuro,spike,Ix,number+5);}
/********************************************************************************/

/*************************突触第七颗虚拟计算单元******************************/
if(number+6<=boxnum[3])
{inhi_syna(neuro_syn,neuro,spike,Ix,number+6);}
/********************************************************************************/

/*************************突触第八颗虚拟计算单元******************************/
if(number+7<=boxnum[3])
{inhi_syna(neuro_syn,neuro,spike,Ix,number+7);}
/********************************************************************************/

/*************************突触第九颗虚拟计算单元******************************/
if(number+8<=boxnum[3])
{inhi_syna(neuro_syn,neuro,spike,Ix,number+8);}
/********************************************************************************/

/*************************突触第十颗虚拟计算单元******************************/
if(number+9<=boxnum[3])
{inhi_syna(neuro_syn,neuro,spike,Ix,number+9);}
/********************************************************************************/

}
