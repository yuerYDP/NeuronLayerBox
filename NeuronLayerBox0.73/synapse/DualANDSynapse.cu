#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

__device__ static void dual_and_syna(struct synapse *syn, struct axon *neuro,unsigned char *spike,struct neuron_I *Ix,int number)
{
  float w;
  float v;
  int I;
  float f;
  float g_NMDA;
  float g_AMPA;
  float t_f;
  float t_r;
  float gpeak;
  float timeN;
  float p;             //max g(t)
  float dt;
  float g_scale;
  unsigned long int prespike;
  float T,s,R;
  float t_max,g2_peak;

  w=syn[number].w;
  timeN=syn[number].timeN;
  v=neuro[syn[number].postsynapse_number].v;
  timeN++;

  prespike=(syn[number].axon<<1) | spike[syn[number].persynapse_number];
  if(((prespike>>syn[number].axon_delay)&0x01)==1) //判断突触前神经元是否有信号
  {
    timeN=0;
  }
  syn[number].axon=prespike;

  //NMDA
  t_f=syn[number].t1_f;
  t_r=syn[number].t1_r;
  gpeak=syn[number].g1;
  g_scale=1/(1+expf((-v-25)/12.5));
  s=syn[number].s1;
  if(timeN*tau<t_r){T=1.0/t_r;}
  else{T=0;}
  if(t_r==t_f)
  {
    s=s+tau*(T*(1-s)-s/t_f);
    g_NMDA=g_scale*gpeak*s*g1_peak;
  }
  else
  {
    R=syn[number].R1;
    t_max=(t_f*t_r/(t_f-t_r))*log(t_f/t_r);
    g2_peak=1+(t_r/t_f)/(2*R_max*expf((t_r-t_max)/t_r));
    R=R+tau*((1.0-R)*T-R/t_r);
    s=s+tau*((t_f+t_r)/t_f)*((2.0/t_r)*(1.0-s)*R-(s/t_f));
    g_NMDA=g_scale*gpeak*s*g2_peak;
  }
  syn[number].s1=s;
  syn[number].R1=R;

  //AMPA
  t_f=syn[number].t2_f;
  t_r=syn[number].t2_r;
  gpeak=syn[number].g2;
  s=syn[number].s2;
  if(timeN*tau<t_r){T=1.0/t_r;}
  else{T=0;}
  if(t_r==t_f)
  {
    s=s+tau*(T*(1-s)-s/t_f);
    g_AMPA=gpeak*s*g1_peak;
  }
  else
  {
    R=syn[number].R2;
    t_max=(t_f*t_r/(t_f-t_r))*log(t_f/t_r);
    g2_peak=1+(t_r/t_f)/(2*R_max*expf((t_r-t_max)/t_r));
    R=R+tau*((1.0-R)*T-R/t_r);
    s=s+tau*((t_f+t_r)/t_f)*((2.0/t_r)*(1.0-s)*R-(s/t_f));
    g_AMPA=gpeak*s*g2_peak;
  }
  syn[number].s2=s;
  syn[number].R2=R;

  I=g_NMDA*(0-v)+w*g_AMPA*(0-v);
  atomicAdd(&Ix[syn[number].postsynapse_number].I,I); //  Ix[syn[number].postsynapse_number]+=I_AMPA+I_NMDA;

if(spike[syn[number].postsynapse_number]==1)//判断突触后神经元是否有信号
{
  f=0.5;
}
else if(timeN<=1)
{
  f=-1.5;
}
else if(timeN<=126)
{
  f=(timeN-1)*tau*0.02-0.5;
}
else
{
  f=0;
}
w=w+lr*tau*g_AMPA*f*f*(g_AMPA*f*(-2)+1-w);
//  if(w<0.5){printf("%f\n",w); }

syn[number].w=w;
syn[number].timeN=timeN;
}

__global__ static void Dual_AND_Synapse(struct synapse *syn, struct axon *neuro,unsigned char *spike,struct neuron_I *Ix,int *boxnum,int *THREAD_SYN,int *BLOCK_SYN)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
	int number=(bid * THREAD_SYN[0] + tid)*10;


/*************************突触第一颗虚拟计算单元******************************/
  if(number+0<=boxnum[0])
  {dual_and_syna(syn,neuro,spike,Ix,number+0);}

/********************************************************************************/

/*************************突触第二颗虚拟计算单元******************************/
if(number+1<=boxnum[0])
{dual_and_syna(syn,neuro,spike,Ix,number+1);}

/********************************************************************************/

/*************************突触第三颗虚拟计算单元******************************/
if(number+2<=boxnum[0])
{dual_and_syna(syn,neuro,spike,Ix,number+2);}

/********************************************************************************/

/*************************突触第四颗虚拟计算单元******************************/
if(number+3<=boxnum[0])
{dual_and_syna(syn,neuro,spike,Ix,number+3);}

/********************************************************************************/

/*************************突触第五颗虚拟计算单元******************************/
if(number+4<=boxnum[0])
{dual_and_syna(syn,neuro,spike,Ix,number+4);}

/********************************************************************************/

/*************************突触第六颗虚拟计算单元******************************/
if(number+5<=boxnum[0])
{dual_and_syna(syn,neuro,spike,Ix,number+5);}

/********************************************************************************/

/*************************突触第七颗虚拟计算单元******************************/
if(number+6<=boxnum[0])
{dual_and_syna(syn,neuro,spike,Ix,number+6);}

/********************************************************************************/

/*************************突触第八颗虚拟计算单元******************************/
if(number+7<=boxnum[0])
{dual_and_syna(syn,neuro,spike,Ix,number+7);}

/********************************************************************************/

/*************************突触第九颗虚拟计算单元******************************/
if(number+8<=boxnum[0])
{dual_and_syna(syn,neuro,spike,Ix,number+8);}

/********************************************************************************/

/*************************突触第十颗虚拟计算单元******************************/
if(number+9<=boxnum[0])
{dual_and_syna(syn,neuro,spike,Ix,number+9);}

/********************************************************************************/
}
